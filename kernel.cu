#define THREADS_PER_BLOCK 128

__global__ void
spmv(int m, int nnz, const int* M_rows, const int* M_cols, const float* M_vals, const float* V_in, float* V_out)
{
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    if (row >= m)
        return;

    register float answer = 0.0;
    int lb = M_rows[row],
        ub = M_rows[row+1];

    for(int offset = lb; offset < ub; offset++)
    answer += M_vals[offset] * V_in[ M_cols[offset] ];

    V_out[row] = answer;
}

extern "C" {


#include <hip/hip_runtime.h>
#include <hipsparse.h>

hipsparseStatus_t
my_cusparseScsrmv(hipsparseHandle_t handle, hipsparseOperation_t transA,
    int m, int n, int nnz, float* alpha,
    hipsparseMatDescr_t descrA,
    const float *csrValA,
    const int *csrRowPtrA, const int *csrColIndA,
    const float *x, float* beta,
    float *y ) {

    const int* M_rows = csrRowPtrA;
    const int* M_cols = csrColIndA;
    const float* M_vals = csrValA;
    const float* V_in = x;
    float* V_out = y;

    int blks = (m + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    spmv<<<blks,THREADS_PER_BLOCK>>>(m, nnz, M_rows, M_cols, M_vals, V_in, V_out);

    return HIPSPARSE_STATUS_SUCCESS;
}

} // extern C
